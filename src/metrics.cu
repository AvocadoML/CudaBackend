/*
 * metrics.cu
 *
 *  Created on: Dec 27, 2021
 *      Author: Maciej Kozarzewski
 */

#include <avocado/cuda_backend.h>
#include <avocado/backend/backend_descriptors.hpp>

#include "utilities.hpp"

#include <hip/hip_runtime_api.h>
#include <hip/hip_fp16.h>

#include <cassert>

namespace
{

}

namespace avocado
{
	namespace backend
	{
		avStatus_t cudaMetricFunction(avContextDescriptor_t context, avMetricType_t metricType, const avTensorDescriptor_t outputDesc,
				const avMemoryDescriptor_t outputMem, const avTensorDescriptor_t targetDesc, const avMemoryDescriptor_t targetMem, void *result)
		{
			return AVOCADO_STATUS_NOT_SUPPORTED;
		}

	} /* namespace backend */
} /* namespace avocado */
