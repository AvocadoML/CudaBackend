/*
 * metrics.cu
 *
 *  Created on: Dec 27, 2021
 *      Author: Maciej Kozarzewski
 */

#include <Avocado/cuda_backend.h>
#include <Avocado/backend_descriptors.hpp>

#include "activations.cuh"
#include "utilities.hpp"

#include <hip/hip_runtime_api.h>
#include <hip/hip_fp16.h>

#include <cassert>

namespace
{

}

namespace avocado
{
	namespace backend
	{
		using namespace BACKEND_NAMESPACE;

		avStatus_t cudaMetricFunction(avContextDescriptor_t context, avMetricType_t metricType, const avTensorDescriptor_t outputDesc,
				const avMemoryDescriptor_t outputMem, const avTensorDescriptor_t targetDesc, const avMemoryDescriptor_t targetMem, void *result)
		{
			getContext(context).setDevice();
			return AVOCADO_STATUS_NOT_SUPPORTED;
		}

	} /* namespace backend */
} /* namespace avocado */
