#include "hip/hip_runtime.h"
/*
 * descriptors.cu
 *
 *  Created on: Dec 22, 2021
 *      Author: Maciej Kozarzewski
 */

#include <Avocado/cuda_backend.h>
#include <Avocado/backend_descriptors.hpp>

#include "utilities.hpp"

#include <cstring>
#include <memory>
#include <hip/hip_runtime_api.h>
#include <hipblas.h>

namespace
{
	using namespace avocado::backend;

	template<typename T>
	__global__ void kernel_setall(T *ptr, int length, T value)
	{
		for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < length; i += gridDim.x * blockDim.x)
			ptr[i] = value;
	}
	template<typename T>
	avStatus_t setall_launcher(hipStream_t stream, T *dst, int dstSize, const void *value)
	{
		const int length = dstSize / sizeof(T);
		dim3 blockDim(256);
		dim3 gridDim = gridSize<1024>(length, blockDim.x);

		T v;
		std::memcpy(&v, value, sizeof(T));
		kernel_setall<<<gridDim, blockDim, 0, stream>>>(dst, length, v);
		return checkForErrors();
	}
}

namespace avocado
{
	namespace backend
	{
		using namespace avocado::backend::BACKEND_NAMESPACE;

		avStatus_t cudaCreateMemoryDescriptor(avMemoryDescriptor_t *result, avDeviceIndex_t deviceIndex, av_int64 sizeInBytes)
		{
			return create_descriptor<MemoryDescriptor>(result, sizeInBytes, deviceIndex);
		}
		avStatus_t cudaCreateMemoryView(avMemoryDescriptor_t *result, const avMemoryDescriptor_t desc, av_int64 sizeInBytes, av_int64 offsetInBytes)
		{
			return create_descriptor<MemoryDescriptor>(result, getMemory(desc), sizeInBytes, offsetInBytes);
		}
		avStatus_t cudaDestroyMemoryDescriptor(avMemoryDescriptor_t desc)
		{
			return destroy_descriptor<MemoryDescriptor>(desc);
		}
		avStatus_t cudaSetMemory(avContextDescriptor_t context, avMemoryDescriptor_t dst, av_int64 dstOffset, av_int64 dstSize, const void *pattern,
				av_int64 patternSize)
		{
			if (not same_device_type(context, dst))
				return AVOCADO_STATUS_DEVICE_TYPE_MISMATCH;
			if (getPointer(dst) == nullptr)
			{
				if (dstSize != 0)
					return AVOCADO_STATUS_BAD_PARAM;
				else
					return AVOCADO_STATUS_SUCCESS;
			}
			try
			{
				getContext(context).setDevice();
				if (pattern == nullptr)
				{
					hipError_t status = hipGetLastError();
					if (isDefault(context))
						status = hipMemset(getPointer<int8_t>(dst) + dstOffset, 0, dstSize);
					else
						status = hipMemsetAsync(getPointer<int8_t>(dst) + dstOffset, 0, dstSize, getContext(context).getStream());
					return convertStatus(status);
				}

				if (dstSize % patternSize != 0 or dstOffset % patternSize != 0)
					return AVOCADO_STATUS_BAD_PARAM;
				switch (patternSize)
				{
					case 1:
						return setall_launcher(getContext(context).getStream(), getPointer<int8_t>(dst) + dstOffset, dstSize, pattern);
					case 2:
						return setall_launcher(getContext(context).getStream(), getPointer<int16_t>(dst) + dstOffset / 2, dstSize, pattern);
					case 4:
						return setall_launcher(getContext(context).getStream(), getPointer<int32_t>(dst) + dstOffset / 4, dstSize, pattern);
					case 8:
						return setall_launcher(getContext(context).getStream(), getPointer<int2>(dst) + dstOffset / 8, dstSize, pattern);
					case 16:
						return setall_launcher(getContext(context).getStream(), getPointer<int4>(dst) + dstOffset / 16, dstSize, pattern);
					default:
						return AVOCADO_STATUS_BAD_PARAM;
				}
			} catch (std::exception &e)
			{
			}
			return AVOCADO_STATUS_INTERNAL_ERROR;
		}
		avStatus_t cudaCopyMemory(avContextDescriptor_t context, avMemoryDescriptor_t dst, av_int64 dstOffset, const avMemoryDescriptor_t src,
				av_int64 srcOffset, av_int64 count)
		{
			if (not same_device_type(context, dst, src))
				return AVOCADO_STATUS_DEVICE_TYPE_MISMATCH;
			try
			{
				getContext(context).setDevice();
				bool is_direct_copy_possible;
				cudaIsCopyPossible(getMemory(src).device(), getMemory(dst).device(), &is_direct_copy_possible);
				if (is_direct_copy_possible) // can use peer-to-peer copy
				{
					hipError_t status;
					if (isDefault(context))
						status = hipMemcpy(getPointer<int8_t>(dst) + dstOffset, getPointer<int8_t>(src) + srcOffset, count, hipMemcpyDeviceToDevice);
					else
						status = hipMemcpyAsync(getPointer<int8_t>(dst) + dstOffset, getPointer<int8_t>(src) + srcOffset, count, hipMemcpyDeviceToDevice,
								getContext(context).getStream());
					return convertStatus(status);
				}
				else // must use intermediate host buffer
				{
					std::unique_ptr<int8_t[]> buffer = std::make_unique<int8_t[]>(count);
					avStatus_t status = cudaCopyMemoryToHost(context, buffer.get(), src, srcOffset, count);
					if (status != AVOCADO_STATUS_SUCCESS)
						return status;
					status = cudaCopyMemoryFromHost(context, dst, dstOffset, buffer.get(), count);
					if (status != AVOCADO_STATUS_SUCCESS)
						return status;
				}
			} catch (std::exception &e)
			{
				return AVOCADO_STATUS_INTERNAL_ERROR;
			}
			return AVOCADO_STATUS_SUCCESS;
		}
		avStatus_t cudaCopyMemoryToHost(avContextDescriptor_t context, void *dst, const avMemoryDescriptor_t src, av_int64 srcOffset, av_int64 bytes)
		{
			if (dst == nullptr)
				return AVOCADO_STATUS_BAD_PARAM;
			hipError_t status;
			try
			{
				getContext(context).setDevice();
				if (isDefault(context))
					status = hipMemcpy(dst, getPointer<int8_t>(src) + srcOffset, bytes, hipMemcpyDeviceToHost);
				else
					status = hipMemcpyAsync(dst, getPointer<int8_t>(src) + srcOffset, bytes, hipMemcpyDeviceToHost, getContext(context).getStream());
			} catch (std::exception &e)
			{
				return AVOCADO_STATUS_INTERNAL_ERROR;
			}
			return convertStatus(status);
		}
		avStatus_t cudaCopyMemoryFromHost(avContextDescriptor_t context, avMemoryDescriptor_t dst, av_int64 dstOffset, const void *src, av_int64 bytes)
		{
			if (src == nullptr)
				return AVOCADO_STATUS_BAD_PARAM;
			hipError_t status;
			try
			{
				getContext(context).setDevice();
				if (isDefault(context))
					status = hipMemcpy(getPointer<int8_t>(dst) + dstOffset, src, bytes, hipMemcpyHostToDevice);
				else
					status = hipMemcpyAsync(getPointer<int8_t>(dst) + dstOffset, src, bytes, hipMemcpyHostToDevice, getContext(context).getStream());
			} catch (std::exception &e)
			{
				return AVOCADO_STATUS_INTERNAL_ERROR;
			}
			return convertStatus(status);
		}
		avStatus_t cudaPageLock(void *ptr, av_int64 count)
		{
			if (ptr == nullptr)
				return AVOCADO_STATUS_BAD_PARAM;
			hipError_t status = hipHostRegister(ptr, count, 0);
			return convertStatus(status);;
		}
		avStatus_t cudaPageUnlock(void *ptr)
		{
			if (ptr == nullptr)
				return AVOCADO_STATUS_BAD_PARAM;
			hipError_t status = hipHostUnregister(ptr);
			return convertStatus(status);;
		}

		int cudaGetNumberOfDevices()
		{
			return getNumberOfDevices();
		}

		avStatus_t cudaCreateContextDescriptor(avContextDescriptor_t *result, avDeviceIndex_t deviceIndex)
		{
			return create_descriptor<ContextDescriptor>(result, deviceIndex);
		}
		avStatus_t cudaDestroyContextDescriptor(avContextDescriptor_t desc)
		{
			return destroy_descriptor<ContextDescriptor>(desc);
		}
		avContextDescriptor_t cudaGetDefaultContext(avDeviceIndex_t deviceIndex)
		{
			if (deviceIndex >= 0 and deviceIndex < cudaGetNumberOfDevices())
				return createDescriptor(deviceIndex, ContextDescriptor::descriptor_type);
			else
				return static_cast<avContextDescriptor_t>(-1);
		}
		avStatus_t cudaSynchronizeWithContext(avContextDescriptor_t context)
		{
			try
			{
				getContext(context).setDevice();
				hipError_t status = hipStreamSynchronize(getContext(context).getStream());
				if (status != hipSuccess)
					return AVOCADO_STATUS_INTERNAL_ERROR;
			} catch (std::exception &e)
			{
				return AVOCADO_STATUS_INTERNAL_ERROR;
			}
			return AVOCADO_STATUS_SUCCESS;
		}
		avStatus_t cudaIsContextReady(avContextDescriptor_t context, bool *result)
		{
			if (result == nullptr)
				return AVOCADO_STATUS_BAD_PARAM;
			try
			{
				getContext(context).setDevice();
				hipError_t status = hipStreamQuery(getContext(context).getStream());
				if (status == hipSuccess)
					result[0] = true;
				else
				{
					if (status == hipErrorNotReady)
						result[0] = false;
					else
					{
						result[0] = false;
						return convertStatus(status);
					}
				}
			} catch (std::exception &e)
			{
				return AVOCADO_STATUS_INTERNAL_ERROR;
			}
			return AVOCADO_STATUS_SUCCESS;
		}

		avStatus_t cudaCreateTensorDescriptor(avTensorDescriptor_t *result)
		{
			return create_descriptor<TensorDescriptor>(result, std::initializer_list<int> { }, AVOCADO_DTYPE_UNKNOWN);
		}
		avStatus_t cudaDestroyTensorDescriptor(avTensorDescriptor_t desc)
		{
			return destroy_descriptor<TensorDescriptor>(desc);
		}
		avStatus_t cudaSetTensorDescriptor(avTensorDescriptor_t desc, avDataType_t dtype, int nbDims, const int dimensions[])
		{
			if (nbDims < 0 or nbDims > AVOCADO_MAX_TENSOR_DIMENSIONS)
				return AVOCADO_STATUS_BAD_PARAM;
			if (dimensions == nullptr and nbDims != 0)
				return AVOCADO_STATUS_BAD_PARAM;

			try
			{
				getTensor(desc).set(dtype, nbDims, dimensions);
			} catch (std::exception &e)
			{
				return AVOCADO_STATUS_INTERNAL_ERROR;
			}
			return AVOCADO_STATUS_SUCCESS;
		}
		avStatus_t cudaGetTensorDescriptor(avTensorDescriptor_t desc, avDataType_t *dtype, int *nbDims, int dimensions[])
		{
			try
			{
				getTensor(desc).get(dtype, nbDims, dimensions);
			} catch (std::exception &e)
			{
				return AVOCADO_STATUS_INTERNAL_ERROR;
			}
			return AVOCADO_STATUS_SUCCESS;
		}

		avStatus_t cudaCreateConvolutionDescriptor(avConvolutionDescriptor_t *result)
		{
			return create_descriptor<ConvolutionDescriptor>(result);
		}
		avStatus_t cudaDestroyConvolutionDescriptor(avConvolutionDescriptor_t desc)
		{
			return destroy_descriptor<ConvolutionDescriptor>(desc);
		}
		avStatus_t cudaSetConvolutionDescriptor(avConvolutionDescriptor_t desc, avConvolutionMode_t mode, int nbDims, const int padding[], const int strides[],
				const int dilation[], int groups, const void *paddingValue)
		{
			try
			{
				getConvolution(desc).set(mode, nbDims, padding, strides, dilation, groups, paddingValue);
			} catch (std::exception &e)
			{
				return AVOCADO_STATUS_INTERNAL_ERROR;
			}
			return AVOCADO_STATUS_SUCCESS;
		}
		avStatus_t cudaGetConvolutionDescriptor(avConvolutionDescriptor_t desc, avConvolutionMode_t *mode, int *nbDims, int padding[], int strides[],
				int dilation[], int *groups, void *paddingValue)
		{
			try
			{
				getConvolution(desc).get(mode, nbDims, padding, strides, dilation, groups, paddingValue);
			} catch (std::exception &e)
			{
				return AVOCADO_STATUS_INTERNAL_ERROR;
			}
			return AVOCADO_STATUS_SUCCESS;
		}

		avStatus_t cudaCreateOptimizerDescriptor(avOptimizerDescriptor_t *result)
		{
			return create_descriptor<OptimizerDescriptor>(result);
		}
		avStatus_t cudaDestroyOptimizerDescriptor(avOptimizerDescriptor_t desc)
		{
			return destroy_descriptor<OptimizerDescriptor>(desc);
		}
		avStatus_t cudaSetOptimizerDescriptor(avOptimizerDescriptor_t desc, avOptimizerType_t type, av_int64 steps, double learningRate,
				const double coefficients[], const bool flags[])
		{
			try
			{
				getOptimizer(desc).set(type, steps, learningRate, coefficients, flags);
			} catch (std::exception &e)
			{
				return AVOCADO_STATUS_INTERNAL_ERROR;
			}
			return AVOCADO_STATUS_SUCCESS;
		}
		avStatus_t cudaGetOptimizerDescriptor(avOptimizerDescriptor_t desc, avOptimizerType_t *type, av_int64 *steps, double *learningRate,
				double coefficients[], bool flags[])
		{
			try
			{
				getOptimizer(desc).get(type, steps, learningRate, coefficients, flags);
			} catch (std::exception &e)
			{
				return AVOCADO_STATUS_INTERNAL_ERROR;
			}
			return AVOCADO_STATUS_SUCCESS;
		}
		avStatus_t cudaGetOptimizerWorkspaceSize(avOptimizerDescriptor_t desc, const avTensorDescriptor_t wDesc, av_int64 *result)
		{
			if (result == nullptr)
				return AVOCADO_STATUS_BAD_PARAM;
			try
			{
				getOptimizer(desc).get_workspace_size(result, getTensor(wDesc));
			} catch (std::exception &e)
			{
				return AVOCADO_STATUS_INTERNAL_ERROR;
			}
			return AVOCADO_STATUS_SUCCESS;
		}

	} /* namespace backend */
} /* namespace avocado */

