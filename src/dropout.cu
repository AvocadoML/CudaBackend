/*
 * dropout.cu
 *
 *  Created on: Dec 27, 2021
 *      Author: Maciej Kozarzewski
 */

#include <CudaBackend/cuda_backend.h>
#include <backend_descriptors.hpp>

#include "utilities.hpp"

#include <hip/hip_runtime_api.h>
#include <hip/hip_fp16.h>

#include <cassert>

namespace avocado
{
	namespace backend
	{
		avStatus_t cudaDropoutForward(avContextDescriptor_t context, const avDropoutDescriptor_t config, const avTensorDescriptor_t xDesc,
				const avMemoryDescriptor_t xMem, const avTensorDescriptor_t yDesc, avMemoryDescriptor_t yMem, avMemoryDescriptor_t states)
		{
			return AVOCADO_STATUS_NOT_SUPPORTED;
		}

		DLL_PUBLIC avStatus_t cudaDropoutBackward(avContextDescriptor_t context, const avDropoutDescriptor_t config, const avTensorDescriptor_t dyDesc,
				const avMemoryDescriptor_t dyMem, const avTensorDescriptor_t dxDesc, avMemoryDescriptor_t dxMem, const avTensorDescriptor_t states)
		{
			return AVOCADO_STATUS_NOT_SUPPORTED;
		}
	} /* namespace backend */
} /* namespace avocado */
