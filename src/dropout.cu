/*
 * dropout.cu
 *
 *  Created on: Dec 27, 2021
 *      Author: Maciej Kozarzewski
 */

#include <CudaBackend/cuda_backend.h>
#include <backend_descriptors.hpp>

#include "utilities.hpp"

#include <hip/hip_runtime_api.h>
#include <hip/hip_fp16.h>

#include <cassert>

namespace avocado
{
	namespace backend
	{
		avStatus_t cudaDropoutForward(avContextDescriptor_t context, const avDropoutDescriptor_t config, const avTensorDescriptor_t xDesc,
				const avMemoryDescriptor_t xMem, const avTensorDescriptor_t yDesc, avMemoryDescriptor_t yMem, avMemoryDescriptor_t states)
		{
			cuda::getContext(context).setDevice();
			return AVOCADO_STATUS_NOT_SUPPORTED;
		}

		avStatus_t cudaDropoutBackward(avContextDescriptor_t context, const avDropoutDescriptor_t config, const avTensorDescriptor_t dyDesc,
				const avMemoryDescriptor_t dyMem, const avTensorDescriptor_t dxDesc, avMemoryDescriptor_t dxMem, const avTensorDescriptor_t states)
		{
			cuda::getContext(context).setDevice();
			return AVOCADO_STATUS_NOT_SUPPORTED;
		}
	} /* namespace backend */
} /* namespace avocado */
