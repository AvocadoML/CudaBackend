/*
 * batchnorm.cu
 *
 *  Created on: Dec 27, 2021
 *      Author: Maciej Kozarzewski
 */

#include <avocado/cuda_backend.h>
#include <avocado/backend/backend_descriptors.hpp>

#include "utilities.hpp"

#include <hip/hip_runtime_api.h>
#include <hip/hip_fp16.h>

#include <cassert>

namespace
{

}

namespace avocado
{
	namespace backend
	{
		avStatus_t cudaAffineForward(avContextDescriptor_t context, avActivationType_t activation, const avTensorDescriptor_t wDesc,
				const avMemoryDescriptor_t wMem, const avTensorDescriptor_t bDesc, const avMemoryDescriptor_t bMem, const void *alpha,
				const avTensorDescriptor_t xDesc, const avMemoryDescriptor_t xMem, const void *beta, const avTensorDescriptor_t yDesc,
				avMemoryDescriptor_t yMem)
		{
			return AVOCADO_STATUS_NOT_SUPPORTED;
		}

		avStatus_t cudaBatchNormInference(avContextDescriptor_t context, avActivationType_t activation, const void *alpha, const avTensorDescriptor_t xDesc,
				const avMemoryDescriptor_t xMem, const void *beta, const avTensorDescriptor_t yDesc, avMemoryDescriptor_t yMem,
				const avTensorDescriptor_t scaleBiasMeanVarDesc, const avMemoryDescriptor_t scaleMem, const avMemoryDescriptor_t biasMem,
				const avMemoryDescriptor_t meanMem, const avMemoryDescriptor_t varianceMem, double epsilon)
		{
			return AVOCADO_STATUS_NOT_SUPPORTED;
		}

		avStatus_t cudaBatchNormForward(avContextDescriptor_t context, avActivationType_t activation, const void *alpha, const avTensorDescriptor_t xDesc,
				const avMemoryDescriptor_t xMem, const void *beta, const avTensorDescriptor_t yDesc, avMemoryDescriptor_t yMem,
				const avTensorDescriptor_t scaleBiasMeanVarDesc, const avMemoryDescriptor_t scaleMem, const avMemoryDescriptor_t biasMem,
				avMemoryDescriptor_t meanMem, avMemoryDescriptor_t varianceMem, double epsilon)
		{
			return AVOCADO_STATUS_NOT_SUPPORTED;
		}

		avStatus_t cudaBatchNormBackward(avContextDescriptor_t context, avActivationType_t activation, const void *alpha, const avTensorDescriptor_t xDesc,
				const avMemoryDescriptor_t xMem, const avTensorDescriptor_t yDesc, const avMemoryDescriptor_t yMem, const void *beta,
				const avTensorDescriptor_t dxDesc, avMemoryDescriptor_t dxMem, const avTensorDescriptor_t dyDesc, avMemoryDescriptor_t dyMem,
				const avTensorDescriptor_t scaleMeanVarDesc, const avMemoryDescriptor_t scaleMem, const avMemoryDescriptor_t meanMem,
				const avMemoryDescriptor_t varianceMem, double epsilon)
		{
			return AVOCADO_STATUS_NOT_SUPPORTED;
		}

		avStatus_t cudaBatchNormUpdate(avContextDescriptor_t context, const void *alpha, const avTensorDescriptor_t xDesc, const avMemoryDescriptor_t xMem,
				const avTensorDescriptor_t dyDesc, const avMemoryDescriptor_t dyMem, const void *beta, const avTensorDescriptor_t scaleBiasDesc,
				avMemoryDescriptor_t scaleUpdateMem, avMemoryDescriptor_t biasUpdateMem, const avMemoryDescriptor_t meanMem,
				const avMemoryDescriptor_t varianceMem, double epsilon)
		{
			return AVOCADO_STATUS_NOT_SUPPORTED;
		}
	} /* namespace backend */
} /* namespace avocado */
