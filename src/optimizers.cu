#include "hip/hip_runtime.h"
/*
 * optimizers.cu
 *
 *  Created on: Dec 27, 2021
 *      Author: Maciej Kozarzewski
 */

#include <CudaBackend/cuda_backend.h>
#include <backend_descriptors.hpp>

#include "activations.cuh"
#include "utilities.hpp"

#include <hip/hip_runtime_api.h>
#include <hip/hip_fp16.h>

#include <cassert>

namespace
{
	using namespace avocado::backend;

	template<typename T>
	__device__ T round_small_to_zero(T x)
	{
		if (x > -scalar_eps<T>() and x < scalar_eps<T>())
			return scalar_zero<T>();
		else
			return x;
	}

	template<typename T>
	__global__ void kernel_learn_sgd(T *weight, const T *update, T *momentum, avSize_t elements, T learning_rate, T beta1, bool use_momentum, bool use_nesterov,
			T alpha, T beta)
	{
		for (uint32_t i = blockIdx.x * blockDim.x + threadIdx.x; i < elements; i += gridDim.x * blockDim.x)
		{
			T tmp;
			if (use_momentum)
			{
				momentum[i] = beta1 * momentum[i] - learning_rate * update[i];
				if (use_nesterov)
					tmp = beta1 * momentum[i] - learning_rate * update[i];
				else
					tmp = momentum[i];
			}
			else
				tmp = -learning_rate * update[i];
			weight[i] = round_small_to_zero(alpha * tmp + beta * weight[i]);
		}
	}
	template<typename T>
	__global__ void kernel_learn_adam(T *weight, const T *update, T *momentum, T *variance, avSize_t elements, T learning_rate, T beta1, T beta2, T alpha,
			T beta)
	{
		for (uint32_t i = blockIdx.x * blockDim.x + threadIdx.x; i < elements; i += gridDim.x * blockDim.x)
		{
			momentum[i] = momentum[i] * beta1 + update[i] * (scalar_one<T>() - beta1);
			variance[i] = variance[i] * beta2 + square(update[i]) * (scalar_one<T>() - beta2);
			T tmp = -momentum[i] * learning_rate / sqrt(variance[i] + scalar_eps<T>());
			weight[i] = round_small_to_zero(alpha * tmp + beta * weight[i]);
		}
	}

	avStatus_t launcher_sgd(const cuda::ContextDescriptor &context, const cuda::OptimizerDescriptor &config, const void *alpha, const void *beta,
			const cuda::TensorDescriptor &wDesc, cuda::MemoryDescriptor &wMem, const cuda::MemoryDescriptor &dwMem, cuda::MemoryDescriptor& workspace)
	{
		const avSize_t elements = wDesc.volume();
		const bool use_momentum = config.flags[0];
		const bool use_nesterov = config.flags[1];
		if (use_momentum)
		{
			if (workspace.size() < elements * cuda::dataTypeSize(wDesc.dtype()))
				return AVOCADO_STATUS_INTERNAL_ERROR;
		}

		dim3 blockDim(256);
		dim3 gridDim = gridSize<1024>(elements, blockDim.x);
		hipStream_t stream = context.getStream();

		switch (wDesc.dtype())
		{
			case AVOCADO_DTYPE_FLOAT32:
			{
				const float _alpha = cuda::getAlphaValue(alpha);
				const float _beta = cuda::getBetaValue(beta);
				const float beta1 = config.coef[0];
				const float learning_rate = config.learning_rate;
				float *momentum = use_momentum ? workspace.data<float>() : nullptr;
				kernel_learn_sgd<<<gridDim, blockDim, 0, stream>>>(wMem.data<float>(), dwMem.data<float>(), momentum, elements, learning_rate, beta1,
						use_momentum, use_nesterov, _alpha, _beta);
				break;
			}
			case AVOCADO_DTYPE_FLOAT64:
			{
				const double _alpha = cuda::getAlphaValue<double>(alpha);
				const double _beta = cuda::getBetaValue<double>(beta);
				const double beta1 = config.coef[0];
				const double learning_rate = config.learning_rate;
				double *momentum = use_momentum ? workspace.data<double>() : nullptr;
				kernel_learn_sgd<<<gridDim, blockDim, 0, stream>>>(wMem.data<double>(), dwMem.data<double>(), momentum, elements, learning_rate, beta1,
						use_momentum, use_nesterov, _alpha, _beta);
				break;
			}
			default:
				return AVOCADO_STATUS_UNSUPPORTED_DATATYPE;
		}
		return checkForErrors();
	}
	avStatus_t launcher_adam(const cuda::ContextDescriptor & context, const cuda::OptimizerDescriptor& config, const void *alpha, const void *beta,
			const cuda::TensorDescriptor & wDesc, cuda::MemoryDescriptor & wMem, const cuda::MemoryDescriptor & dwMem, cuda::MemoryDescriptor & workspace)
	{
		const avSize_t elements = wDesc.volume();

		if (workspace.size() < 2 * elements * cuda::dataTypeSize(wDesc.dtype()))
			return AVOCADO_STATUS_INTERNAL_ERROR;

		dim3 blockDim(256);
		dim3 gridDim = gridSize<1024>(elements, blockDim.x);
		hipStream_t stream = context.getStream();

		switch (wDesc.dtype())
		{
			case AVOCADO_DTYPE_FLOAT32:
			{
				const float _alpha = cuda::getAlphaValue(alpha);
				const float _beta = cuda::getBetaValue(beta);
				const float beta1 = config.coef[0];
				const float beta2 = config.coef[1];
				const float learning_rate = config.learning_rate;
				kernel_learn_adam<<<gridDim, blockDim, 0, stream>>>(wMem.data<float>(), dwMem.data<float>(), workspace.data<float>(),
						workspace.data<float>() + elements, elements, learning_rate, beta1, beta2, _alpha, _beta);
				break;
			}
			case AVOCADO_DTYPE_FLOAT64:
			{
				const double _alpha = cuda::getAlphaValue<double>(alpha);
				const double _beta = cuda::getBetaValue<double>(beta);
				const double beta1 = config.coef[0];
				const double beta2 = config.coef[1];
				const double learning_rate = config.learning_rate;
				kernel_learn_adam<<<gridDim, blockDim, 0, stream>>>(wMem.data<double>(), dwMem.data<double>(), workspace.data<double>(),
						workspace.data<double>() + elements, elements, learning_rate, beta1, beta2, _alpha, _beta);
				break;
			}
			default:
				return AVOCADO_STATUS_UNSUPPORTED_DATATYPE;
		}
		return checkForErrors();
	}
}

namespace avocado
{
	namespace backend
	{
		avStatus_t cudaOptimizerLearn(avContextDescriptor_t context, const avOptimizerDescriptor_t config, const void *alpha, const avTensorDescriptor_t dwDesc,
				const avTensorDescriptor_t dwMem, const void *beta, const avTensorDescriptor_t wDesc, avMemoryDescriptor_t wMem, avMemoryDescriptor_t workspace)
		{
			cuda::getContext(context).setDevice();
			switch (cuda::getOptimizer(config).type)
			{
				case AVOCADO_OPTIMIZER_SGD:
					return launcher_sgd(cuda::getContext(context), cuda::getOptimizer(config), alpha, beta, cuda::getTensor(wDesc), cuda::getMemory(wMem),
							cuda::getMemory(dwMem), cuda::getMemory(workspace));
				case AVOCADO_OPTIMIZER_ADAM:
					return launcher_adam(cuda::getContext(context), cuda::getOptimizer(config), alpha, beta, cuda::getTensor(wDesc), cuda::getMemory(wMem),
							cuda::getMemory(dwMem), cuda::getMemory(workspace));
				default:
					return AVOCADO_STATUS_BAD_PARAM;
			}
			return AVOCADO_STATUS_SUCCESS;
		}

	} /* namespace backend */
} /* namespace avocado */
