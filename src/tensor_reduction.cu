#include "hip/hip_runtime.h"
/*
 * convert.cu
 *
 *  Created on: Sep 16, 2020
 *      Author: Maciej Kozarzewski
 */

#include <CudaBackend/cuda_backend.h>
#include <backend_descriptors.hpp>

#include "utilities.hpp"
#include "activations.cuh"

#include <cstring>

#include <hip/hip_fp16.h>
#include <hip/hip_runtime_api.h>

namespace
{
	using namespace avocado::backend;

	template<typename T>
	struct limits
	{
		__device__ T max() const noexcept
		{
			return zero<T>();
		}
	};
	template<>
	struct limits<half>
	{
		__device__ float max() const noexcept
		{
			return 65504;
		}
	};
	template<>
	struct limits<float>
	{
		__device__ float max() const noexcept
		{
			return 3.40282346638528859811704183484516925e+38f;
		}
	};
	template<>
	struct limits<double>
	{
		__device__ double max() const noexcept
		{
			return 1.79769313486231570814527423731704357e+308;
		}
	};

	template<typename T>
	class ReduceAdd
	{
		T acc = zero<T>();
	public:
		__device__ ReduceAdd() = default;
		__device__ void accumulate(T x) noexcept
		{
			acc += x;
		}
		__device__ void combine_partial(ReduceAdd other) noexcept
		{
			acc += other.acc;
		}
		__device__ ReduceAdd& operator=(T value) noexcept
		{
			acc = value;
			return *this;
		}
		__device__ operator T() const noexcept
		{
			return acc;
		}
	};
	template<typename T>
	class ReduceMul
	{
		T acc = one<T>();
	public:
		__device__ ReduceMul() = default;
		__device__ void accumulate(T x) noexcept
		{
			acc *= x;
		}
		__device__ void combine_partial(ReduceMul other) noexcept
		{
			acc *= other.acc;
		}
		__device__ ReduceMul& operator=(T value) noexcept
		{
			acc = value;
			return *this;
		}
		__device__ operator T() const noexcept
		{
			return acc;
		}
	};
	template<typename T>
	class ReduceMin
	{
		T acc = limits<T>().max();
	public:
		__device__ ReduceMin() = default;
		__device__ void accumulate(T x) noexcept
		{
			this->acc = min(this->acc, x);
		}
		__device__ void combine_partial(ReduceMin other) noexcept
		{
			this->acc = min(this->acc, other.acc);
		}
		__device__ ReduceMin& operator=(T value) noexcept
		{
			acc = value;
			return *this;
		}
		__device__ operator T() const noexcept
		{
			return acc;
		}
	};
	template<typename T>
	class ReduceMax
	{
		T acc = -limits<T>().max();
	public:
		__device__ ReduceMax() = default;
		__device__ void accumulate(T x) noexcept
		{
			acc = max(acc, x);
		}
		__device__ void combine_partial(ReduceMax other) noexcept
		{
			acc = max(acc, other.acc);
		}
		__device__ ReduceMax& operator=(T value) noexcept
		{
			acc = value;
			return *this;
		}
		__device__ operator T() const noexcept
		{
			return acc;
		}
	};
	template<typename T>
	class ReduceAMax
	{
		T acc = zero<T>();
	public:
		__device__ ReduceAMax() = default;
		__device__ void accumulate(T x) noexcept
		{
			acc = max(acc, abs(x));
		}
		__device__ void combine_partial(ReduceAMax other) noexcept
		{
			acc = max(acc, other.acc);
		}
		__device__ ReduceAMax& operator=(T value) noexcept
		{
			acc = value;
			return *this;
		}
		__device__ operator T() const noexcept
		{
			return acc;
		}
	};
	template<typename T>
	class ReduceNorm1
	{
		T acc = zero<T>();
	public:
		__device__ ReduceNorm1() = default;
		__device__ void accumulate(T x) noexcept
		{
			acc += abs(x);
		}
		__device__ void combine_partial(ReduceNorm1 other) noexcept
		{
			acc += other.acc;
		}
		__device__ ReduceNorm1& operator=(T value) noexcept
		{
			acc = value;
			return *this;
		}
		__device__ operator T() const noexcept
		{
			return acc;
		}
	};
	template<typename T>
	class ReduceNorm2
	{
		T acc = zero<T>();
	public:
		__device__ ReduceNorm2() = default;
		__device__ void accumulate(T x) noexcept
		{
			acc += square(x);
		}
		__device__ void combine_partial(ReduceNorm2 other) noexcept
		{
			acc += other.acc;
		}
		__device__ ReduceNorm2& operator=(T value) noexcept
		{
			acc = value;
			return *this;
		}
		__device__ operator T() const noexcept
		{
			return acc;
		}
	};
	template<typename T>
	class ReduceMulNoZeros
	{
		T acc = one<T>();
	public:
		__device__ ReduceMulNoZeros() = default;
		__device__ void accumulate(T x) noexcept
		{
			if (x != zero<T>())
				acc *= x;
		}
		__device__ void combine_partial(ReduceMulNoZeros other) noexcept
		{
			acc *= other.acc;
		}
		__device__ ReduceMulNoZeros& operator=(T value) noexcept
		{
			acc = value;
			return *this;
		}
		__device__ operator T() const noexcept
		{
			return acc;
		}
	};

	template<class Acc>
	__device__ void block_reduce_linear(Acc *ptr) noexcept
	{
		assert(ispow2(blockDim.x));
		for (unsigned int i = blockDim.x / 2; i >= 1; i /= 2) // sum results stored in temporary array
		{
			if (threadIdx.x < i)
				ptr[threadIdx.x].combine_partial(ptr[threadIdx.x + i]);
			__syncthreads();
		}
	}
	template<class Acc, typename T>
	__global__ void kernel_reduce_linear_1(T *dst, const T* src, unsigned int elements)
	{
		__shared__ Acc storage[1024];

		Acc acc;
		for (unsigned int i = blockIdx.x * blockDim.x + threadIdx.x; i < elements; i += blockDim.x * gridDim.x)
			acc.accumulate(src[i]);

		storage[threadIdx.x] = acc;

		__syncthreads();
		block_reduce_linear(storage);
		if (threadIdx.x == 0)
			dst[blockIdx.x] = storage[0];
	}
	template<class Acc, typename T, typename U = T>
	__global__ void kernel_reduce_linear_2(T *dst, const T* src, U alpha, U beta)
	{
		__shared__ Acc storage[1024];
		storage[threadIdx.x] = src[threadIdx.x];
		__syncthreads();
		block_reduce_linear(storage);
		if (threadIdx.x == 0)
		{
			T tmp = alpha * static_cast<T>(storage[0]);
			if (beta != zero<U>())
				tmp += beta * dst[0];
			dst[0] = tmp;
		}
	}

	template<class Acc>
	__device__ void block_reduce_broadcasted(Acc *ptr) noexcept
	{
		for (int i = 16; i >= 1; i /= 2) // sum results stored in temporary array
		{
			if (threadIdx.y < i)
				ptr[threadIdx.y * 32 + threadIdx.x].combine_partial(ptr[(i + threadIdx.y) * 32 + threadIdx.x]);
			__syncthreads();
		}
	}
	template<class Acc, typename T>
	__global__ void kernel_reduce_broadcasted_1(T *dst, const T* src, unsigned int first_dim, unsigned int last_dim)
	{
		__shared__ Acc storage[32 * 32];
		for (unsigned int j = blockIdx.x * blockDim.x; j < last_dim; j += blockDim.x * gridDim.x)
		{
			unsigned int idx = j + threadIdx.x;

			Acc acc;
			if (idx < last_dim)
			{
				for (unsigned int i = blockIdx.y * blockDim.y + threadIdx.y; i < first_dim; i += blockDim.y * gridDim.y)
					acc.accumulate(src[i * last_dim + idx]);
			}
			storage[threadIdx.y * 32 + threadIdx.x] = acc;

			__syncthreads();
			block_reduce_broadcasted(storage);
			if (threadIdx.y == 0 and idx < last_dim)
				dst[blockIdx.y * last_dim + idx] = storage[0 * 32 + threadIdx.x];
		}
	}
	template<class Acc, typename T, typename U = T>
	__global__ void kernel_reduce_broadcasted_2(T *dst, const T* src, U alpha, U beta, unsigned int first_dim, unsigned int last_dim)
	{
		__shared__ Acc storage[32 * 32];
		for (unsigned int j = blockIdx.x * blockDim.x; j < last_dim; j += blockDim.x * gridDim.x)
		{
			unsigned int idx = j + threadIdx.x;

			Acc acc;
			if (idx < last_dim)
			{
				for (unsigned int i = blockIdx.y * blockDim.y + threadIdx.y; i < first_dim; i += blockDim.y * gridDim.y)
					acc.combine_partial(reinterpret_cast<const Acc*>(src)[i * last_dim + idx]);
			}
			storage[threadIdx.y * 32 + threadIdx.x] = acc;

			__syncthreads();
			block_reduce_broadcasted(storage);
			if (threadIdx.y == 0 and idx < last_dim)
			{
				T tmp = alpha * static_cast<T>(storage[0 * 32 + threadIdx.x]);
				if (beta != zero<U>())
					tmp += beta * dst[blockIdx.y * last_dim + idx];
				dst[blockIdx.y * last_dim + idx] = tmp;
			}
		}
	}

	template<class Op, typename T, typename U = T>
	void helper_reduce_tensor(hipStream_t stream, T* output, const T *input, const U alpha, const U beta, cuda::BroadcastedDimensions dimensions, T* workspace)
	{
		if (dimensions.last == 1) // output is a single element
		{
			const int partial_results = 64; // must be power of 2
			kernel_reduce_linear_1<Op, T> <<<partial_results, 1024, 0, stream>>>(workspace, input, dimensions.first);
			kernel_reduce_linear_2<Op, T, U> <<<1, partial_results, 0, stream>>>(output, workspace, alpha, beta);
		}
		else
		{
			dim3 blockDim(32, 32);
			dim3 gridDim1(8, 128);
			kernel_reduce_broadcasted_1<Op, T> <<<gridDim1, blockDim, 0, stream>>>(workspace, input, dimensions.first, dimensions.last);

			dim3 gridDim2(8, 1);
			kernel_reduce_broadcasted_2<Op, T, U> <<<gridDim2, blockDim, 0, stream>>>(output, workspace, alpha, beta, dimensions.first, dimensions.last);
		}
	}
	template<typename T, typename U = T>
	avStatus_t launcher_reduce_tensor(hipStream_t stream, T* dst, const T *input, const U alpha, const U beta, cuda::BroadcastedDimensions dimensions,
			avReduceOp_t operation, T* workspace)
	{
		switch (operation)
		{
			case AVOCADO_REDUCE_ADD:
				helper_reduce_tensor<ReduceAdd<T>, T, U>(stream, dst, input, alpha, beta, dimensions, workspace);
				break;
			case AVOCADO_REDUCE_MUL:
				helper_reduce_tensor<ReduceMul<T>, T, U>(stream, dst, input, alpha, beta, dimensions, workspace);
				break;
			case AVOCADO_REDUCE_MIN:
				helper_reduce_tensor<ReduceMin<T>, T, U>(stream, dst, input, alpha, beta, dimensions, workspace);
				break;
			case AVOCADO_REDUCE_MAX:
				helper_reduce_tensor<ReduceMax<T>, T, U>(stream, dst, input, alpha, beta, dimensions, workspace);
				break;
			case AVOCADO_REDUCE_AMAX:
				helper_reduce_tensor<ReduceAMax<T>, T, U>(stream, dst, input, alpha, beta, dimensions, workspace);
				break;
			case AVOCADO_REDUCE_AVG:
				helper_reduce_tensor<ReduceAdd<T>, T, U>(stream, dst, input, alpha / dimensions.first, beta, dimensions, workspace);
				break;
			case AVOCADO_REDUCE_NORM1:
				helper_reduce_tensor<ReduceNorm1<T>, T, U>(stream, dst, input, alpha, beta, dimensions, workspace);
				break;
			case AVOCADO_REDUCE_NORM2:
				helper_reduce_tensor<ReduceNorm2<T>, T, U>(stream, dst, input, alpha, beta, dimensions, workspace);
				break;
			case AVOCADO_REDUCE_MUL_NO_ZEROS:
				helper_reduce_tensor<ReduceMulNoZeros<T>, T, U>(stream, dst, input, alpha, beta, dimensions, workspace);
				break;
			default:
				return AVOCADO_STATUS_BAD_PARAM;
		}
		return checkForErrors();
	}
}

namespace avocado
{
	namespace backend
	{

		avStatus_t cudaReduceTensor(avContextDescriptor_t context, avReduceOp_t operation, const void *alpha, const avTensorDescriptor_t aDesc,
				const avMemoryDescriptor_t aMem, const void *beta, const avTensorDescriptor_t cDesc, avMemoryDescriptor_t cMem)
		{
			cuda::BroadcastedDimensions dimensions = cuda::getBroadcastDimensions(cuda::getTensor(aDesc), cuda::getTensor(cDesc));
			hipStream_t stream = cuda::getContext(context).getStream();
			cuda::getContext(context).setDevice();

			switch (cuda::getTensor(aDesc).dtype())
			{
//				case AVOCADO_DTYPE_FLOAT16:
//					return launcher_reduce_tensor(stream, cuda::getPointer<half>(cMem), cuda::getPointer<half>(aMem), cuda::getAlphaValue(alpha), getBetaValue(beta),
//							dimensions, operation, cuda::getContext(context).getWorkspace().data<half>());
				case AVOCADO_DTYPE_FLOAT32:
					return launcher_reduce_tensor(stream, cuda::getPointer<float>(cMem), cuda::getPointer<float>(aMem), cuda::getAlphaValue(alpha),
							cuda::getBetaValue(beta), dimensions, operation, cuda::getContext(context).getWorkspace().data<float>());
				case AVOCADO_DTYPE_FLOAT64:
					return launcher_reduce_tensor(stream, cuda::getPointer<double>(cMem), cuda::getPointer<double>(aMem), cuda::getAlphaValue<double>(alpha),
							cuda::getBetaValue<double>(beta), dimensions, operation, cuda::getContext(context).getWorkspace().data<double>());
				default:
					return AVOCADO_STATUS_UNSUPPORTED_DATATYPE;
			}
		}
	} /* namespace backend */
} /* namespace avocado */
