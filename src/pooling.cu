/*
 * pooling.cu
 *
 *  Created on: Dec 27, 2021
 *      Author: Maciej Kozarzewski
 */

#include <Avocado/cuda_backend.h>
#include <Avocado/backend_descriptors.hpp>

#include "utilities.hpp"

#include <hip/hip_runtime_api.h>
#include <hip/hip_fp16.h>

#include <cassert>

namespace
{

}

namespace avocado
{
	namespace backend
	{
		using namespace BACKEND_NAMESPACE;

		avStatus_t cudaPoolingForward(avContextDescriptor_t context, const avPoolingDescriptor_t config, const void *alpha, const avTensorDescriptor_t xDesc,
				const avMemoryDescriptor_t xMem, const void *beta, const avTensorDescriptor_t yDesc, avMemoryDescriptor_t yMem)
		{
			getContext(context).setDevice();
			return AVOCADO_STATUS_NOT_SUPPORTED;
		}

		avStatus_t cudaPoolingBackward(avContextDescriptor_t context, const avPoolingDescriptor_t config, const void *alpha, const avTensorDescriptor_t xDesc,
				const avMemoryDescriptor_t xMem, const avTensorDescriptor_t dyDesc, const avMemoryDescriptor_t dyMem, const void *beta,
				const avTensorDescriptor_t dxDesc, avMemoryDescriptor_t dxMem)
		{
			getContext(context).setDevice();
			return AVOCADO_STATUS_NOT_SUPPORTED;
		}

	} /* namespace backend */
} /* namespace avocado */
